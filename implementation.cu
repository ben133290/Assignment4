/*
============================================================================
Filename    : algorithm.c
Author      : Benedikt Heuser
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

// GPU Optimized function
__global__ void kernel(double *input, double *output, int length) {
    // Calculate global indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    

    if (i > 0 && i < length - 1 && j > 0 && j < length - 1) {
        // Compute the convolution
        output[i * length + j] =
            (input[(i - 1) * length + (j - 1)] + input[(i - 1) * length + j] + input[(i - 1) * length + (j + 1)] +
             input[i * length + (j - 1)] + input[i * length + j] + input[i * length + (j + 1)] +
             input[(i + 1) * length + (j - 1)] + input[(i + 1) * length + j] + input[(i + 1) * length + (j + 1)]) /
            9.0;
    }

    // Set special values
    int m = length/2;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        output[m * length + m] = 1000.0;
        output[(m-1) * length + m] = 1000.0;
        output[(m-1) * length + m-1] = 1000.0;
        output[m * length + m-1] = 1000.0;
    }


    /*
    if (i == length / 2 - 1 && j == length / 2 - 1)
        output[i * length + j] = 1000.0;

    if (i == length / 2 && j == length / 2 - 1)
        output[i * length + j] = 1000.0;

    if (i == length / 2 - 1 && j == length / 2)
        output[i * length + j] = 1000.0;

    if (i == length / 2 && j == length / 2)
        output[i * length + j] = 1000.0;
        */
    
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double *gpu_input, *gpu_output;
    
    hipMalloc((void**)&gpu_input, length * length * sizeof(double));
    hipMalloc((void**)&gpu_output, length * length * sizeof(double));

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipMemcpy(gpu_input, input, length * length * sizeof(double), hipMemcpyHostToDevice);

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    // Copy array from host to device <- I think this is misplaced but it was in the original ???
    hipEventRecord(comp_start);

    /* GPU calculation goes here */
    dim3 threadsPerBlock(32, 32);
    int blockSide = length/32;
    if (length % 32 != 0) {blockSide++;}
    dim3 numOfBlocks(length/32 + 1, length/32 + 1);
    for (int n = 0; n < iterations; n++) {

        kernel<<<numOfBlocks, threadsPerBlock>>>(gpu_input, gpu_output, length);

        if (n < iterations - 1) {
            double* temp = gpu_input;
            gpu_input = gpu_output;
            gpu_output = temp;
        }
    }

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);



    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy(output, gpu_output, length * length * sizeof(double), hipMemcpyDeviceToHost);
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(gpu_input);
    hipFree(gpu_output);

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}